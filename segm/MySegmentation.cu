#include "hip/hip_runtime.h"
/*
 * This file is part of https://github.com/martinruenz/maskfusion
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>
 */

#include "MySegmentation.cuh"
#include "../cuda/segmentation.cuh"
#include "../cuda/cudafuncs.cuh"
#include <opencv2/imgproc/imgproc.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <memory>
#include <algorithm>

MySegmentation::MySegmentation(int w, int h, const CameraModel& cameraIntrinsics) :
    minMappedComponentSize(160) {

    floatEdgeMap.create(h, w);
    floatBuffer.create(h, w);

    ucharBuffer.create(h, w);
    binaryEdgeMap.create(h, w);

    cv8UC1Buffer.create(h, w, CV_8UC1);
    cvLabelComps.create(h, w, CV_32S);
    cvLabelEdges.create(h, w, CV_32S);
    semanticIgnoreMap = cv::Mat::zeros(h, w, CV_8UC1);

    this->cameraIntrinsics = cameraIntrinsics;
    vertexMap.create(h*3,w);
    normalMap.create(h*3,w);
    depthMapMetric.create(h,w);
    depthMapMetricFiltered.create(h,w);
    rgb.create(h,w);
}

MySegmentation::~MySegmentation(){}

cv::Mat MySegmentation::performSegmentation(FrameData& frame)
{    
    cv::Mat result;
    const int& width = frame.depth.cols;
    const int& height = frame.depth.rows;
    const size_t total = frame.depth.total();
    result = cv::Mat::zeros(height, width, CV_8UC1);
    const int nMasks = int(frame.classIDs.size());
    const float maxRelSizeNew = 0.4;
    const float minRelSizeNew = 0.07;
    const size_t minNewMaskPixels = minRelSizeNew * total;
    const size_t maxNewMaskPixels = maxRelSizeNew * total;

    // Prepare data (vertex/depth/... maps)
    
    computeLookups(frame);
    computeGeometricSegmentationMap(vertexMap, normalMap, floatEdgeMap, weightDistance, weightConvexity);
    
    // Perform geometric segmentation

    
    DeviceArray2D<float>& edgeMap = floatEdgeMap;
    
    thresholdMap(edgeMap, binaryEdgeMap, threshold);
    morphGeometricSegmentationMap(binaryEdgeMap,ucharBuffer, morphEdgeRadius, morphEdgeIterations);
    invertMap(binaryEdgeMap,ucharBuffer);
    ucharBuffer.download(cv8UC1Buffer.data, ucharBuffer.cols());

#if true // FIXME: segmentation debugging

    cv::imshow("Segmentation", cv8UC1Buffer);

#endif

    // Build use ignore map
    if(nMasks)
    {
        for(size_t i=0; i<total; i++)
        {
            if(frame.classIDs[frame.mask.data[i]] == personClassID)
            {
                semanticIgnoreMap.data[i] = 255;
                cv8UC1Buffer.data[i] = 0;
            }
            else
            {
                semanticIgnoreMap.data[i] = 0;
            }
        }
        //cv::compare(frame.mask, cv::Scalar(...), semanticIgnoreMap, CV_CMP_EQ);
    }
    else
    {
        for(size_t i=0; i<total; i++)
        {
            if(semanticIgnoreMap.data[i])
                cv8UC1Buffer.data[i] = 0;
        }
    }

    // Run connected-components on segmented map
    cv::Mat statsComp, centroidsComp;
    int nComponents = cv::connectedComponentsWithStats(cv8UC1Buffer, cvLabelComps, statsComp, centroidsComp, 4);
    

    // Todo, this can be faster! (GPU?)
    if(removeEdges)
    {
        const bool remove_small_components = true;
        const int small_components_threshold = 50;
        const int removeEdgeIterations = 5;
        
        
        auto checkNeighbor = [&, this](int y, int x, int& n, float d) {
            n = this->cvLabelComps.at<int>(y,x);
            if (n != 0 && std::fabs(frame.depth.at<float>(y,x)-d) < 0.008 && statsComp.at<int>(n, 4) > small_components_threshold)
            {
                return true;
            }
            return false;
        };
        for (int i = 0; i < removeEdgeIterations; ++i)
        {
            cv::Mat r;
            cvLabelComps.copyTo(r);
            for (int y = 1; y < height-1; ++y) // TODO reduce index computations here
            {
                for (int x = 1; x < width-1; ++x)
                {
                    int& c = r.at<int>(y,x);
//                    statsComp.at<int>(c, 4);
                    float d = frame.depth.at<float>(y,x);

                    if(c==0 || (remove_small_components && statsComp.at<int>(c, 4) < small_components_threshold))
                    {
                        int c2;
                        if(checkNeighbor(y-1,x-1,c2,d)) { c = c2; continue; }
                        if(checkNeighbor(y-1,x,c2,d)) { c = c2; continue; }
                        if(checkNeighbor(y-1,x+1,c2,d)) { c = c2; continue; }
                        if(checkNeighbor(y,x-1,c2,d)) { c = c2; continue; }
                        if(checkNeighbor(y,x+1,c2,d)) { c = c2; continue; }
                        if(checkNeighbor(y+1,x-1,c2,d)) { c = c2; continue; }
                        if(checkNeighbor(y+1,x,c2,d)) { c = c2; continue; }
                        if(checkNeighbor(y+1,x+1,c2,d)) { c = c2; continue; }
                    }
                }
            }
            cvLabelComps = r;
        }
    }

    // Assign mask to each component
    // Replace edges and persons with 255
//    mapComponentToMask[0] = 255; // Edges

    // Group components that belong to the same mask
    
    std::vector<int> mapComponentToMask(nComponents, 0); // By default, components are mapped to background (maskid==0)
    std::vector<int> maskComponentPixels(nMasks, 0); // Number of pixels per mask
    std::vector<BoundingBox> maskComponentBoxes(nMasks);
    cv::Mat compMaskOverlap(nComponents,nMasks,CV_32SC1, cv::Scalar(0));

    if(nMasks)
    {

        // Compute component-mask overlap
        for (size_t i = 0; i < total; ++i)
        {
            const unsigned char& mask_val = frame.mask.data[i];
            const int& comp_val = cvLabelComps.at<int>(i);
            //assert(frame.classIDs.size() > mask_val);
            //if(mask_val != 255)
            compMaskOverlap.at<int>(comp_val,mask_val)++;
        }

        // Compute mapping
        const float overlap_threshold = 0.65;
        for (int c = 1; c < nComponents; ++c)
        {
            int& csize = statsComp.at<int>(c, 4);
            if(csize > minMappedComponentSize)
            {
                int t = overlap_threshold * csize;
                for (int m = 1; m < nMasks; ++m)
                {
                    if(compMaskOverlap.at<int>(c,m) > t)
                    {
                        mapComponentToMask[c] = m;
                        maskComponentPixels[m] += statsComp.at<int>(c, 4);
                        maskComponentBoxes[m].mergeLeftTopWidthHeight(statsComp.at<int>(c, 0),
                                                                      statsComp.at<int>(c, 1),
                                                                      statsComp.at<int>(c, 2),
                                                                      statsComp.at<int>(c, 3));
                    }
                }
            }
            else
            {
                // Map tiny component to ignored
                //mapComponentToMask[c] = 255;

                // Map tiny component to background
                mapComponentToMask[c] = 0;
            }
        }
    }

    for (size_t i = 0; i < total; ++i)
        result.data[i] = mapComponentToMask[cvLabelComps.at<int>(i)];
    
    // FIX HACK
    for(size_t i=0; i<total; i++)
        if(semanticIgnoreMap.data[i])
            result.data[i] = 255;
    
    if(removeEdgeIslands && nMasks)
    {
        // Remove "edge islands" within masks
        cv::threshold(result, cv8UC1Buffer, 254, 255, cv::THRESH_TOZERO); // THRESH_BINARY is equivalent here
        cv::Mat statsEdgeComp, centroidsEdgeComp;
        int nEdgeComp = cv::connectedComponentsWithStats(cv8UC1Buffer, cvLabelEdges, statsEdgeComp, centroidsEdgeComp, 4);
        //cv::imshow("edge labels", mapLabelToColorImage(cvLabelEdges));



        for (int ec = 1; ec < nEdgeComp; ++ec)
        {
            for (int m = 1; m < nMasks; ++m)
            {
                BoundingBox bb = BoundingBox::fromLeftTopWidthHeight(statsEdgeComp.at<int>(ec,0),
                                                                     statsEdgeComp.at<int>(ec,1),
                                                                     statsEdgeComp.at<int>(ec,2),
                                                                     statsEdgeComp.at<int>(ec,3));
                if(maskComponentBoxes[m].includes(bb))
                {
                    //std::cout << "mask " << m << " fully contains edge-component " << ec << std::endl;
                    int x1 = std::max(bb.left+1,1);
                    int x2 = std::min(bb.right, width-2);
                    int y1 = std::max(bb.top+1, 1);
                    int y2 = std::min(bb.bottom, height-2);
                    bool doBreak = false;
                    for (int y = y1; y <= y2; ++y) {
                        for (int x = x1; x <= x2; ++x) {
                            const int& le = cvLabelEdges.at<int>(y,x-1); // TODO this can be a bit faster
                            const int& te = cvLabelEdges.at<int>(y-1,x);
                            const int& ce = cvLabelEdges.at<int>(y,x);
                            const unsigned char& lm = result.at<unsigned char>(y,x-1);
                            const unsigned char& tm = result.at<unsigned char>(y-1,x);
                            const unsigned char& cm = result.at<unsigned char>(y,x);
                            if( (le!=ec && ce==ec && lm!=m) ||
                                    (le==ec && ce!=ec && cm!=m) ||
                                    (te!=ec && ce==ec && tm!=m) ||
                                    (te==ec && ce!=ec && cm!=m)) {
                                doBreak = true;
                                break;
                            }
                        }
                        if(doBreak) break;
                    }
                    if(doBreak) break;

                    // This can only happen once, replace component
                    for (int y = bb.top; y <= bb.bottom; ++y) {
                        for (int x = bb.left; x <= bb.right; ++x) {
                            if (cvLabelEdges.at<int>(y,x)==ec){
                                result.at<unsigned char>(y,x) = m;
                                //islands.at<unsigned char>(y,x) = 255;
                            }
                        }
                    }
                }
            }
        }
    }
    
    return result;
}

void MySegmentation::computeLookups(FrameData& frame)
{
    // Copy OpenGL depth texture for CUDA use
    // textureDepthMetric->cudaMap();
    // hipArray* depthTexturePtr = textureDepthMetric->getCudaArray();
    // hipMemcpy2DFromArray(depthMapMetric.ptr(0), depthMapMetric.step(), depthTexturePtr, 0, 0, depthMapMetric.colsBytes(), depthMapMetric.rows(),
    //                       hipMemcpyDeviceToDevice);
    // textureDepthMetric->cudaUnmap();

    // textureRGB->cudaMap();
    // hipArray* rgbTexturePtr = textureRGB->getCudaArray();
    // hipMemcpy2DFromArray(rgb.ptr(0), rgb.step(), rgbTexturePtr, 0, 0, rgb.colsBytes(), rgb.rows(), hipMemcpyDeviceToDevice);
    // textureRGB->cudaUnmap();
    rgb.upload(frame.rgb.data, frame.rgb.cols * sizeof(unsigned char) * 4, frame.rgb.rows, frame.rgb.cols);
    depthMapMetric.upload(frame.depth.data, frame.depth.cols * sizeof(float), frame.depth.rows, frame.depth.cols);

    // Custom filter for depth map
    bilateralFilter(rgb, depthMapMetric, depthMapMetricFiltered, bilatSigmaRadius, 0, bilatSigmaDepth, bilatSigmaColor, bilatSigmaLocation);
    //    hipArray* debugMapPtr = debugMap->getCudaArray();
    //    hipMemcpy2DToArray(debugMapPtr, 0, 0, depthMapMetricFiltered.ptr(0), depthMapMetricFiltered.step(), depthMapMetricFiltered.colsBytes(), depthMapMetricFiltered.rows(), hipMemcpyDeviceToDevice);

    // Generate buffers for vertex and normal maps
    createVMap(cameraIntrinsics, depthMapMetricFiltered, vertexMap, 999.0f);
    createNMap(vertexMap, normalMap);

#if true // FIXME: normal map debugging

    // get normal map result
    cv::Mat sample(frame.depth.rows, frame.depth.cols, CV_32FC3);
    normalMap.download(sample.data, sample.cols * sizeof(float));

    // match normal map with OpenCV format
    cv::Mat relocte(frame.depth.rows, frame.depth.cols, CV_32FC3);
    int step = frame.depth.rows * frame.depth.cols;
    for (int i = 0; i < step; i++)
    {
        ((float*)relocte.data)[3 * i + 0] = ((float*)sample.data)[i + step * 2];
        ((float*)relocte.data)[3 * i + 1] = ((float*)sample.data)[i + step * 1];
        ((float*)relocte.data)[3 * i + 2] = ((float*)sample.data)[i + step * 0];
    }
    
    // display normal map image
    cv::imshow("Normal Map", relocte * 0.5f + 0.5f);

#endif

}
